#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include ""

#include "emicp.h"

struct updateA_functor
{
	float sigma;
	updateA_functor(float _sigma)
	{
		sigma = _sigma;
	}
	__host__ __device__ float operator()(const float3& p1, const float3& p2)
	{
		float tmp0, tmp1, tmp2;
		tmp0 = p1.x - p2.x;
		tmp1 = p1.y - p2.y;
		tmp2 = p1.z - p2.z;
		tmp0 = tmp0 * tmp0 + tmp1 * tmp1 + tmp2 * tmp2;
		tmp0 /= sigma;
		return expf(-tmp0);
	}
};

void EMICP::updateA( Mat& A, const Mat& objSet, 
	const Mat& modSet, bool withCuda )
{
	
	if (withCuda)
	{
		Mat tmpObj = convertMat(objSet);
		Mat tmpMod = convertMat(modSet);
		cuda_updateA(A, tmpObj, tmpMod);
	}
	else
	{
		int rowsA = objSet.rows;
		int colsA = modSet.rows;

		float3* arr_obj = new float3[rowsA];
		memcpy(arr_obj, (float3*)objSet.data, rowsA * sizeof(float3));
		float3* arr_mod = new float3[colsA];
		memcpy(arr_mod, (float3*)modSet.data, colsA * sizeof(float3));

		try
		{
			thrust::host_vector<float> h_A = 
				thrust::host_vector<float>(rowsA * colsA);
			thrust::host_vector<float3> h_obj(arr_obj, arr_obj + rowsA);
			thrust::host_vector<float3> h_mod(arr_mod, arr_mod + colsA);

			for (int i = 0; i < rowsA; i++)
			{
				thrust::constant_iterator<float3> tmp(h_obj[i]);
				thrust::transform(h_mod.begin(), h_mod.end(), tmp, 
					h_A.begin() + i * colsA, updateA_functor(m_sigma_p2));
			}

			float* h_A_ptr = thrust::raw_pointer_cast(&h_A[0]);
			A = Mat(rowsA, colsA, CV_32FC1);
			memcpy((float*)A.data, h_A_ptr, rowsA * colsA * sizeof(float));
		}
		catch (thrust::system_error e)
		{
			cout << "System Error: " << e.what() << endl;
		}

		delete[] arr_obj;
		delete[] arr_mod;
	}
}

struct normalizeRow_functor
{
	__host__ __device__ float operator()(const float& x, const float& lambda)
	{
		return (float)(x / (lambda + 1e-7));
	}
};

struct sqrt_functor
{
	__host__ __device__ float operator()(const float& x)
	{
		return sqrtf(x);
	}
};

void EMICP::normalizeRows(Mat &mat, const Mat &alpha, bool withCuda, bool withSqrt)
{
	if (withCuda)
	{
		cuda_normalizeRows(mat, alpha, withSqrt);
	}
	else
	{
		int rows = mat.rows;
		int cols = mat.cols;
		float* arr_mat = new float[rows * cols];
		memcpy(arr_mat, (float*)mat.data, rows * cols * sizeof(float));
		float* arr_alpha = new float[rows];
		memcpy(arr_alpha, (float*)alpha.data, rows * sizeof(float));

		try
		{
			thrust::host_vector<float> h_mat(arr_mat, arr_mat + rows * cols);
			thrust::host_vector<float> h_alpha(arr_alpha, arr_alpha + rows);

			for (int i = 0; i < rows; i++)
			{
				thrust::constant_iterator<float> tmp(h_alpha[i]);
				thrust::host_vector<float>::iterator 
					begin = h_mat.begin() + i * cols;
				thrust::transform(begin, begin + cols, 
					tmp, begin, normalizeRow_functor());
			}

			if (withSqrt)
			{
				thrust::transform(h_mat.begin(), h_mat.end(), 
					h_mat.begin(), sqrt_functor());
			}

			float* h_mat_ptr = thrust::raw_pointer_cast(&h_mat[0]);
			memcpy((float*)mat.data, h_mat_ptr, rows * cols * sizeof(float));
		}
		catch (thrust::system_error e)
		{
			cout << "System Error: " << e.what() << endl;
		}

		delete[] arr_mat;
		delete[] arr_alpha;
	}
}

#define BLOCK_SIZE 128

__global__ void kernelUpdateA(PtrStepSz<float> d_mod, PtrStepSz<float> d_obj,
	PtrStep<float> d_A, float sigma_p2)
{
	int x = blockIdx.x * blockDim.x + threadIdx.x;
	int y = blockIdx.y * blockDim.y + threadIdx.y;

	if (y < d_obj.rows && x < d_mod.rows)
	{
		float tmp[3];
		for (int i = 0; i < 3; i++)
		{
			tmp[i] = d_mod(x, i) - d_obj(y, i);
			tmp[i] *= tmp[i];
		}
		tmp[0] += tmp[1];
		tmp[0] += tmp[2];
		tmp[0] /= sigma_p2;
		tmp[0] = expf(-tmp[0]);

		d_A(y, x) = tmp[0];
	}
}

void EMICP::cuda_updateA(Mat &h_A, const Mat &objSet, const Mat &modSet)
{
	assert(objSet.cols == 3 && modSet.cols == 3);

	int rowsA = objSet.rows;
	int colsA = modSet.rows;
	GpuMat d_obj, d_mod;
	d_obj.upload(objSet);
	d_mod.upload(modSet);

	dim3 block(16, 16);
	dim3 grid((colsA + block.x - 1) / block.x, 
		(rowsA + block.y - 1) / block.y);

	GpuMat d_A(rowsA, colsA, CV_32FC1);
	kernelUpdateA<<<grid, block>>>(d_mod, d_obj, d_A, m_sigma_p2);

	d_A.download(h_A);

	d_obj.release();
	d_mod.release();
	d_A.release();
}

__global__ void kernelNormalizeRows(PtrStepSz<float> d_mat, 
	PtrStepSz<float> d_alpha, int withSqrt)
{
	int x = blockIdx.x * blockDim.x + threadIdx.x;
	int y = blockIdx.y * blockDim.y + threadIdx.y;

	if (y < d_mat.rows && x < d_mat.cols)
	{
		if (withSqrt)
		{
			d_mat(y, x) = sqrtf(d_mat(y, x) / (d_alpha(y, 0) + 1e-7));
		}
		else
		{
			d_mat(y, x) = d_mat(y, x) / (d_alpha(y, 0) + 1e-7);
		}
	}
}

void EMICP::cuda_normalizeRows(Mat& mat, 
	const Mat& alpha, bool withSqrt)
{
	int rows = mat.rows;
	int cols = mat.cols;
	GpuMat d_mat, d_alpha;
	d_mat.upload(mat);
	d_alpha.upload(alpha);

	dim3 block(16, 16);
	dim3 grid((cols + block.x - 1) / block.x, 
		(rows + block.y - 1) / block.y);
	kernelNormalizeRows<<<grid, block>>>(d_mat, d_alpha, (int)withSqrt);
	
	d_mat.download(mat);

	d_mat.release();
	d_alpha.release();
}