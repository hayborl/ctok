#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include ""

#include "common.h"

__device__ int cntd[1];

__global__ void cuda_findNeighbor(float* pSet, float3 p, 
	const size_t size)
{
	size_t idx = blockIdx.x * blockDim.x + threadIdx.x;

	if (idx < size && cntd[0] == 0)
	{
		int offset = idx * 3;
		float sum = 0;
		float tmp = p.x - pSet[offset];
		sum += tmp * tmp;
		tmp = p.y - pSet[offset + 1];
		sum += tmp * tmp;
		tmp = p.z - pSet[offset + 2];
		sum += tmp * tmp;
		if (sum < DISTANCE_RANGE)
		{
			cntd[0]++;
		}
	}
}

EXTERN_C void cuda_pushBackPoint(float* pSet1, float* pSet2,  
	const size_t size1, const size_t size2, Mat pointColor,
	vector<Vec3f>& v, vector<Vec3b>& c)
{
	float* pSet1d;
	size_t copySize = size1 * 3 * sizeof(float);
	if (size1 > 0)
	{
		hipMalloc((void**)&pSet1d, copySize);
		hipMemcpy(pSet1d, pSet1, copySize, hipMemcpyHostToDevice);
	}

	Point3f p;
	Vec3b color;
	Vec3f vp;
	for (size_t i = 0; i < size2; i ++/*= SAMPLE_INTERVAL*/)
	{
		size_t off = i * 3;
		p = Point3f(pSet2[off], pSet2[off + 1], pSet2[off + 2]);
		if (p != Point3f(0, 0, 0))
		{
			vp = Vec3f(p);
			vp[2] = -vp[2];
			bool flag = false;

			if (size1 > 0)
			{
				float3 pd;
				pd.x = vp[0];
				pd.y = vp[1];
				pd.z = vp[2];

				int cnt = 0;
// 				hipMemcpyToSymbol(HIP_SYMBOL(cntd), &cnt, sizeof(int));
// 				cuda_findNeighbor<<<size1 / BLOCK_SIZE + 1, 
// 					BLOCK_SIZE>>>(pSet1d, pd, size1);
// 				hipMemcpyFromSymbol(&cnt, HIP_SYMBOL(cntd), sizeof(int));
				
				flag = (cnt > 0);
			}
			if (!flag)
			{
				v.push_back(vp);
				color = pointColor.at<Vec3b>((int)i, 0);
				c.push_back(Vec3b(color[2], color[1], color[0]));
			}
		}
	}

	if (size1 > 0)
	{
		hipFree(pSet1d);
	}
}

char* fileName = "common.cu";

inline void __checkCudaErrors(hipError_t err, 
	const char *file = NULL, const int line = 0)
{
	if (hipSuccess != err)
	{
		fprintf(stderr, "%s(%i) : CUDA Runtime API error %d: %s.\n",
			file, line, (int)err, hipGetErrorString(err));
		exit(EXIT_FAILURE);
	}
}

__global__ void kernelTransform(const float* p, const float* R, 
	const float* T, float* res, int count)
{
	int i = blockIdx.x * blockDim.x + threadIdx.x;

	if (i < count)
	{
		float tp[3];
		tp[0] = p[i * 3 + 0];
		tp[1] = p[i * 3 + 1];
		tp[2] = p[i * 3 + 2];
		float resp[3];
		for(int y = 0; y < 3; y++)
		{
			float tmp = 0;
			for (int x = 0; x < 3; x++)
			{
				int index = y * 3 + x;
				tmp += R[index] * tp[x];
			}
			resp[y] = tmp + T[y];
		}
		res[i * 3 + 0] = resp[0];
		res[i * 3 + 1] = resp[1];
		res[i * 3 + 2] = resp[2];
	}
}

void cuda_transform(const float* p, int count, const float* R, 
	const float* T, float* res)
{
	int size;
	// Load A and B to the device
	float* pd;
	size = count * 3 * sizeof(float);
	__checkCudaErrors(hipMalloc((void**)&pd, size), fileName, 56);
	__checkCudaErrors(hipMemcpy(pd, p, size, hipMemcpyHostToDevice), 
		fileName, 57);
	float* Rd;
	size = 3 * 3 * sizeof(float);
	__checkCudaErrors(hipMalloc((void**)&Rd, size), fileName, 61);
	__checkCudaErrors(hipMemcpy(Rd, R, size, hipMemcpyHostToDevice), 
		fileName, 62);
	// Allocate C on the device
	float* Td;
	size = 3 * 1 * sizeof(float);
	__checkCudaErrors(hipMalloc((void**)&Td, size), fileName, 67);
	__checkCudaErrors(hipMemcpy(Td, T, size, hipMemcpyHostToDevice), 
		fileName, 68);
	float* resd;
	size = count * 3 * sizeof(float);
	__checkCudaErrors(hipMalloc((void**)&resd, size), fileName, 72);
	// Compute the execution configuration assuming
	// the matrix dimensions are multiples of BLOCK_SIZE
	// Launch the device computation
	kernelTransform<<<count / BLOCK_SIZE + 1, BLOCK_SIZE>>>
		(pd, Rd, Td, resd, count);
	// Read C from the device
	__checkCudaErrors(hipMemcpy(res, resd, size, hipMemcpyDeviceToHost), 
		fileName, 80);
	// Free device memory
	hipFree(pd);
	hipFree(Rd);
	hipFree(Td);
	hipFree(resd);
}

EXTERN_C void cuda_transformPointCloud(Mat input, 
	Mat* output, Mat transformMat)
{
	int rows = input.rows;
	int size = rows * 3;
	float* p = new float[size];
	memcpy(p, (float*)input.data, size * sizeof(float));

	Mat MR = transformMat(Rect(0, 0, 3, 3)).clone();
	Mat MT = transformMat(Rect(3, 0, 1, 3)).clone();
	float R[9], T[3];
	memcpy(R, (float*)MR.data, 9 * sizeof(float));
	memcpy(T, (float*)MT.data, 3 * sizeof(float));

	float* res = new float[size];
	cuda_transform(p, rows, R, T, res);
	memcpy((float*)output->data, res, size * sizeof(float));

	delete[] p;
	delete[] res;
}